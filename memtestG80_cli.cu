/*
 * memtestG80_cli.cu
 * Command-line interface frontend for MemtestG80 tester
 *
 * Author: Imran Haque, 2009
 * Copyright 2009, Stanford University
 *
 * This file is licensed under the terms of the LGPL. Please see
 * the COPYING file in the accompanying source distribution for
 * full license terms.
 *
 */

#include <stdlib.h>
#include <stdio.h>
#include <assert.h>
#include <string.h>
#include <ctype.h>
#include "memtestG80_core.h"
#include "ezOptionParser.hpp"

#ifdef OSX
inline size_t strnlen(const char* s,size_t maxlen) {
    size_t i;
    for (i=0;i<maxlen && s[i];i++);
    return i;
}
#endif


bool validateNumeric(const char* str) {
    size_t idlen;
    // Assumes number will not be larger than 10 digits
    if ((idlen=strnlen(str,11))==11)
        return false;
    for (size_t i = 0; i < idlen; i++) {
        if (!isdigit(str[i])) return false;
    }
    return true;
}

void print_usage(void) {
    printf("     -------------------------------------------------------------\n");
    printf("     |                      MemtestG80 v1.00                     |\n");
    //printf("     |             Copyright 2009, Stanford University           |\n");
    printf("     |                                                           |\n");
    printf("     | Usage: memtestG80 [flags] [MB GPU RAM to test] [# iters]  |\n");
    printf("     |                                                           |\n");
    printf("     | Defaults: GPU 0, 128MB RAM, 50 test iterations            |\n");
    printf("     | Amount of tested RAM will be rounded up to nearest 2MB    |\n");
    printf("     -------------------------------------------------------------\n\n");
    printf("      Available flags:\n");
    printf("        --gpu N ,-g N : run test on the Nth (from 0) CUDA GPU\n");
    printf("        --license ,-l : show license terms for this build\n");
    printf("\n");
}

void print_licensing(void) {
    printf("Copyright 2009, Stanford University\n");
    #ifndef WEBCOMM // Don't just #else this! Translation scripts need the ifdef form.
    printf("Licensed under the GNU Library General Public License (LGPL), version 3.0\n");
    printf("Please see the file COPYING in the source distribution for details\n");
    #endif
    printf("\n");
    #if defined(WINDOWS) || defined(WINNV)
    printf("This software incorporates by linkage code from the libintl and libiconv\n");
    printf("libraries, which are covered by the Library GNU Public License, available\n");
    printf("at http://www.gnu.org/licenses/lgpl-3.0.txt\n");
    #endif
    return;
}

int main(int argc,const char** argv) {
    uint megsToTest=128;
    uint maxIters=50;
    int gpuID=0;
    int showLicense = 0;
    hipDeviceProp_t deviceProps;

    print_usage(); 
    ez::ezOptionParser opt;

    opt.add(
        "0", // Default.
        0, // Required?
        1, // Number of args expected.
        0, // Delimiter if expecting multiple args.
        "run test on the Nth (from 0) CUDA GPU", // Help description.
        "--gpu",
        "-g"
    );

    opt.add(
        "", // Default.
        0, // Required?
        0, // Number of args expected.
        0, // Delimiter if expecting multiple args.
        "show license terms for this build\n", // Help description.
        "-l",
        "--license"
    );

    opt.parse(argc, argv);
    std::string lastArg;
    if(opt.isSet("-g"))
        opt.get("-g")->getInt(gpuID);
    if(opt.isSet("-l"))
        opt.get("-g")->getInt(showLicense);
    if(opt.lastArgs.size() == 0) {
        // do nothing, use default settings
    } else if(opt.lastArgs.size() == 2) {
        sscanf(opt.lastArgs[0]->c_str(),"%u",&megsToTest);
        sscanf(opt.lastArgs[1]->c_str(),"%u",&maxIters);
    } else {
        printf("Error: Bad argument for [MB GPU RAM to test] [# iters]");
    }

    if (showLicense) print_licensing();
    
    // Sanity check device ID
    int devCount;
    hipGetDeviceCount(&devCount);
    if (gpuID >= devCount) {
        printf("Error: Specified invalid GPU index (%d); %d CUDA devices present, numbered from zero.\n",gpuID,devCount);
        printf("\nValid CUDA devices:\n");
        for (int i = 0; i < devCount; i++) {
            hipGetDeviceProperties(&deviceProps,i);
            printf("%d: %s\n",i,deviceProps.name);
        }
        exit(2);
    }

    hipGetDeviceProperties(&deviceProps,gpuID);

    // Sanity check for real device
    if ((deviceProps.major == 9999) && (deviceProps.minor == 9999)) {
        printf("Error: No CUDA hardware detected. CUDA device emulator is not supported.\n");
        exit(2);
    }

    // Passed sanity checks, set device to use
    if (hipSetDevice(gpuID) == hipErrorInvalidDevice) {
        printf("Error: Got Invalid Device Error setting %d as active CUDA device.\n",gpuID);
        exit(2);
    }

    // Sanity check RAM size and iteration count
    if (megsToTest <= 0) {
        printf("Error: invalid memory test region size %d MiB\n",megsToTest);
        exit(2);
    }
    if (maxIters <= 0) {
        printf("Error: invalid iteration count %d\n",maxIters);
        exit(2);
    }


    memtestState tester;
    if (!tester.allocate(megsToTest)) {
        printf("Error: unable to allocate %u MiB of GPU memory to test, bailing!\n",megsToTest);
        printf("Error text: %s\n",hipGetErrorString(hipGetLastError()));
        exit(2);
    } else {
        printf("Running %u iterations of tests over %u MB of GPU memory on card %d: %s\n\n",maxIters,tester.size(),gpuID,deviceProps.name);
    }

    // Run bandwidth test
    const unsigned bw_iters = 20;
    printf("Running memory bandwidth test over %u iterations of %u MB transfers...\n",bw_iters,tester.size()/2);
    double bandwidth;
    if (!tester.gpuMemoryBandwidth(bandwidth,tester.size()/2,bw_iters)) {
        printf("\tTest failed!\n");
        bandwidth = 0;
    } else {
        printf("\tEstimated bandwidth %.02f MB/s\n\n",bandwidth);
    }


    uint accumulatedErrors = 0,iterErrors;
    uint errorCounts[15];
    memset(errorCounts,0,15*sizeof(uint));
   
    unsigned int start,end;

    for (uint i = 0; i < maxIters ; i++) { 
        printf("Test iteration %u (GPU %d, %d MiB): %u errors so far\n",i+1,gpuID,tester.size(),accumulatedErrors);
        uint errorCount;
        
        // Moving inversions, 1's and 0's {{{
        errorCount = 0;
        start=getTimeMilliseconds();
        tester.gpuMovingInversionsOnesZeros(errorCount);
        accumulatedErrors += errorCount;
        end=getTimeMilliseconds();
        errorCounts[0] += errorCount;
        printf("\tMoving Inversions (ones and zeros): %u errors (%u ms)\n",errorCount,end-start);
        // }}}
        // Memtest86 walking 8-bit {{{
        errorCount = 0;
        start=getTimeMilliseconds();
        for (uint shift=0;shift<8;shift++){
            tester.gpuWalking8BitM86(iterErrors,shift);
            errorCount += iterErrors;
        }
        end=getTimeMilliseconds();
        accumulatedErrors+=errorCount;
        errorCounts[1] += errorCount;
        printf("\tMemtest86 Walking 8-bit: %u errors (%u ms)\n",errorCount,end-start);
        // }}}
        // True Walking zeros, 8-bit {{{
        errorCount = 0;
        start=getTimeMilliseconds();
        for (uint shift=0;shift<8;shift++){
            tester.gpuWalking8Bit(iterErrors,false,shift);
            errorCount += iterErrors;
        }
        end=getTimeMilliseconds();
        accumulatedErrors+=errorCount;
        errorCounts[2] += errorCount;
        printf("\tTrue Walking zeros (8-bit): %u errors (%u ms)\n",errorCount,end-start);
        // }}}
        // True Walking ones, 8-bit {{{
        errorCount = 0;
        start=getTimeMilliseconds();
        for (uint shift=0;shift<8;shift++){
            tester.gpuWalking8Bit(iterErrors,true,shift);
            errorCount += iterErrors;
        }
        end=getTimeMilliseconds();
        accumulatedErrors+=errorCount;
        errorCounts[3] += errorCount;
        printf("\tTrue Walking ones (8-bit): %u errors (%u ms)\n",errorCount,end-start);
        // }}}
        // Moving inversions, random {{{
        start=getTimeMilliseconds();
        tester.gpuMovingInversionsRandom(errorCount);
        accumulatedErrors += errorCount;
        end=getTimeMilliseconds();
        errorCounts[4] += errorCount;
        printf("\tMoving Inversions (random): %u errors (%u ms)\n",errorCount,end-start);
        // }}}
        // Walking zeros, 32-bit {{{
        errorCount = 0;
        start=getTimeMilliseconds();
        for (uint shift=0;shift<32;shift++){
            tester.gpuWalking32Bit(iterErrors,false,shift);
            errorCount += iterErrors;
        }
        end=getTimeMilliseconds();
        accumulatedErrors+=errorCount;
        errorCounts[5] += errorCount;
        printf("\tMemtest86 Walking zeros (32-bit): %u errors (%u ms)\n",errorCount,end-start);
        // }}}
        // Walking ones, 32-bit {{{
        errorCount = 0;
        start=getTimeMilliseconds();
        for (uint shift=0;shift<32;shift++){
            tester.gpuWalking32Bit(iterErrors,true,shift);
            errorCount += iterErrors;
        }
        end=getTimeMilliseconds();
        accumulatedErrors+=errorCount;
        errorCounts[6] += errorCount;
        printf("\tMemtest86 Walking ones (32-bit): %u errors (%u ms)\n",errorCount,end-start);
        // }}}
        // Random blocks {{{
        if (tester.size() <= 16400){
            start=getTimeMilliseconds();
            tester.gpuRandomBlocks(errorCount,rand());
            accumulatedErrors += errorCount;
            errorCounts[7] += errorCount;
            end=getTimeMilliseconds();
            printf("\tRandom blocks: %u errors (%u ms)\n",errorCount,end-start);
        }
        else
          printf("\tRandom blocks: skipped, see Bug #3569\n");

        // }}}
        // Modulo-20, 32-bit {{{
        errorCount = 0;
        start=getTimeMilliseconds();
        for (uint shift=0;shift<20;shift++){
            tester.gpuModuloX(iterErrors,shift,rand(),20,2);
            errorCount += iterErrors;
        }
        end=getTimeMilliseconds();
        accumulatedErrors+=errorCount;
        errorCounts[8] += errorCount;
        printf("\tMemtest86 Modulo-20: %u errors (%u ms)\n",errorCount,end-start);
        // }}}
        // Logic, 1 iteration {{{
        errorCount = 0;
        start=getTimeMilliseconds();
        tester.gpuShortLCG0(errorCount,1);
        end=getTimeMilliseconds();
        accumulatedErrors += errorCount;
        errorCounts[9] += errorCount;
        printf("\tLogic (one iteration): %u errors (%u ms)\n",errorCount,end-start);
        // }}}
        // Logic, 4 iterations {{{
        errorCount = 0;
        start=getTimeMilliseconds();
        tester.gpuShortLCG0(errorCount,4);
        end=getTimeMilliseconds();
        accumulatedErrors += errorCount;
        errorCounts[10] += errorCount;
        printf("\tLogic (4 iterations): %u errors (%u ms)\n",errorCount,end-start);
        // }}}
       // Logic, shared-memory, 1 iteration {{{
        errorCount = 0;
        start=getTimeMilliseconds();
        tester.gpuShortLCG0Shmem(errorCount,1);
        end=getTimeMilliseconds();
        accumulatedErrors += errorCount;
        errorCounts[11] += errorCount;
        printf("\tLogic (shared memory, one iteration): %u errors (%u ms)\n",errorCount,end-start);
        // }}}
        // Logic, shared-memory, 4 iterations {{{
        errorCount = 0;
        start=getTimeMilliseconds();
        tester.gpuShortLCG0Shmem(errorCount,4);
        end=getTimeMilliseconds();
        accumulatedErrors += errorCount;
        errorCounts[12] += errorCount;
        printf("\tLogic (shared-memory, 4 iterations): %u errors (%u ms)\n",errorCount,end-start);
        // }}}
        
        printf("\n");
    }
    printf("Final error count after %u iterations over %u MiB of GPU memory: %u errors\n",maxIters,tester.size(),accumulatedErrors);
    return (accumulatedErrors != 0);
}
